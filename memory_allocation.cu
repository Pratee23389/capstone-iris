#include "hip/hip_runtime.h"
#include "memory_allocation.h"

__global__ void add(int *d_a, int *d_b, int *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        h_c[i] = d_a[i] + d_b[i];
}

__global__ void sub(int *d_a, int *d_b, int *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        h_c[i] = d_a[i] - d_b[i];
}

__global__ void mult(int *d_a, int *d_b, int *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        h_c[i] = d_a[i] * d_b[i];
}

__global__ void mod(int *d_a, int *d_b, int *h_c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        h_c[i] = d_a[i] % d_b[i];
}

// Allocate host input A (pageable) and B (pinned)
__host__ std::tuple<int *, int *> allocateRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);
    
    int *h_a = (int *)malloc(size);             // Pageable
    int *h_b; hipHostMalloc((void **)&h_b, size);  // Pinned

    for (int i = 0; i < numElements; ++i)
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    return {h_a, h_b};
}

// Read input from CSV
__host__ std::tuple<int *, int *, int> readCsv(std::string filename)
{
    std::vector<int> tempResult;
    std::ifstream file(filename);
    if (!file.is_open()) throw std::runtime_error("Failed to open file");

    std::string line;
    int val;

    getline(file, line);
    std::stringstream ss1(line);
    while (ss1 >> val)
    {
        tempResult.push_back(val);
        if (ss1.peek() == ',') ss1.ignore();
    }

    int numElements = tempResult.size();
    int *h_a = (int *)malloc(numElements * sizeof(int));
    std::copy(tempResult.begin(), tempResult.end(), h_a);
    tempResult.clear();

    getline(file, line);
    std::stringstream ss2(line);
    while (ss2 >> val)
    {
        tempResult.push_back(val);
        if (ss2.peek() == ',') ss2.ignore();
    }

    int *h_b;
    hipHostMalloc((int **)&h_b, numElements * sizeof(int));
    std::copy(tempResult.begin(), tempResult.end(), h_b);
    file.close();

    return {h_a, h_b, numElements};
}

// Allocate device memory
__host__ std::tuple<int *, int *> allocateDeviceMemory(int numElements)
{
    int *d_a, *d_b;
    size_t size = numElements * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    return {d_a, d_b};
}

// Copy host to device memory
__host__ void copyFromHostToDevice(int *h_a, int *h_b, int *d_a, int *d_b, int numElements)
{
    size_t size = numElements * sizeof(int);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
}

// Launch kernel
__host__ void executeKernel(int *d_a, int *d_b, int *h_c, int numElements, int threadsPerBlock, std::string op)
{
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    if (op == "sub")
        sub<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    else if (op == "mult")
        mult<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    else if (op == "mod")
        mod<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);
    else
        add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, h_c, numElements);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Deallocate memory
__host__ void deallocateMemory(int *d_a, int *d_b)
{
    hipFree(d_a);
    hipFree(d_b);
}

// Reset device
__host__ void cleanUpDevice()
{
    hipDeviceReset();
}

// Output result to file
__host__ void outputToFile(std::string partId, int *h_a, int *h_b, int *h_c, int n, std::string op)
{
    std::ofstream file("output-" + partId + ".txt", std::ofstream::app);

    file << "Mathematical Operation: " << op << "\n";
    file << "PartID: " << partId << "\n";
    file << "Input A: ";
    for (int i = 0; i < n; ++i) file << h_a[i] << " ";
    file << "\nInput B: ";
    for (int i = 0; i < n; ++i) file << h_b[i] << " ";
    file << "\nResult: ";
    for (int i = 0; i < n; ++i) file << h_c[i] << " ";
    file << "\n";
    file.close();
}

// Parse command line args
__host__ std::tuple<int, std::string, int, std::string, std::string> parseCommandLineArguments(int argc, char *argv[])
{
    int numElements = 10, threadsPerBlock = 256;
    std::string partId = "test", op = "add", file = "NULL";

    for (int i = 1; i < argc; i += 2)
    {
        std::string opt(argv[i]), val(argv[i+1]);
        if (opt == "-n") numElements = std::stoi(val);
        else if (opt == "-p") partId = val;
        else if (opt == "-t") threadsPerBlock = std::stoi(val);
        else if (opt == "-o") op = val;
        else if (opt == "-f") file = val;
    }

    return {numElements, partId, threadsPerBlock, file, op};
}

// Input generator (random or CSV)
__host__ std::tuple<int *, int *, int> setUpInput(std::string file, int numElements)
{
    if (file != "NULL")
    {
        return readCsv(file);
    }
    else
    {
        auto [a, b] = allocateRandomHostMemory(numElements);
        return {a, b, numElements};
    }
}

int main(int argc, char *argv[])
{
    auto [numElements, partId, threadsPerBlock, file, op] = parseCommandLineArguments(argc, argv);
    auto [h_a, h_b, n] = setUpInput(file, numElements);
    numElements = n;

    int *h_c;
    hipMallocManaged(&h_c, numElements * sizeof(int));  // Unified memory

    auto [d_a, d_b] = allocateDeviceMemory(numElements);
    copyFromHostToDevice(h_a, h_b, d_a, d_b, numElements);
    executeKernel(d_a, d_b, h_c, numElements, threadsPerBlock, op);
    hipDeviceSynchronize();  // Needed for unified memory

    outputToFile(partId, h_a, h_b, h_c, numElements, op);
    deallocateMemory(d_a, d_b);
    hipFree(h_c);
    cleanUpDevice();

    return 0;
}
